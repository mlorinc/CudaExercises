#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <type_traits>
#include <cstdlib>
#include <string.h>
#include <cassert>

constexpr int ARRAY_SIZE = 520000;
constexpr int BLOCK_SIZE = 256;
constexpr int BLOCK_MAX_SIZE = 1024;

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
void cpuScan(T* out, T* vector, int size)
{
	if (size == 0)
	{
		return;
	}

	out[0] = vector[0];
	for (int i = 1; i < size; i++)
	{
		out[i] = out[i - 1] + vector[i];
	}
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__device__ void toScratch(T scratchpad[BLOCK_SIZE], const T* inputVector)
{
	scratchpad[threadIdx.x] = inputVector[blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__device__ int upsweep(T scratchpad[BLOCK_SIZE])
{
	int stride;
	for (stride = 1; stride < blockDim.x; stride *= 2)
	{
		if (threadIdx.x % (stride * 2) == 0)
		{
			scratchpad[threadIdx.x] = scratchpad[threadIdx.x] + scratchpad[threadIdx.x + stride];
		}
		__syncthreads();
	}

	return stride;
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__device__ T upsweepCorrection(T scratchpad[BLOCK_SIZE], T tempSums[BLOCK_SIZE])
{
	const int max = scratchpad[0];
	if (threadIdx.x == 0)
	{
		tempSums[0] = scratchpad[0];
		scratchpad[0] = 0;
	}
	__syncthreads();
	return max;
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__device__ void downsweep(T scratchpad[BLOCK_SIZE], T tempSums[BLOCK_SIZE], int stride)
{
	for (stride /= 2; stride >= 1; stride /= 2)
	{
		if (threadIdx.x % (stride * 2) == 0)
		{
			const int left = tempSums[threadIdx.x] - scratchpad[threadIdx.x + stride];
			tempSums[threadIdx.x] = left;
			tempSums[threadIdx.x + stride] = scratchpad[threadIdx.x + stride];
			scratchpad[threadIdx.x + stride] = scratchpad[threadIdx.x] + left;
		}
		__syncthreads();
	}
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__device__ void downsweepCorrection(T scratchpad[BLOCK_SIZE], T* downsweepVector, int max)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid + 1 < ARRAY_SIZE && threadIdx.x + 1 < blockDim.x)
	{
		downsweepVector[tid] = scratchpad[threadIdx.x + 1];
	}
	else if (tid + 1 == ARRAY_SIZE || threadIdx.x + 1 == blockDim.x)
	{
		downsweepVector[tid] = max;
	}
	else
	{
		// ignore
	}
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void prescanBlockKernel(T* result, T* inputVector, T* maximums)
{
	__shared__ T scratchpad[BLOCK_SIZE];
	__shared__ T tempSums[BLOCK_SIZE];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < ARRAY_SIZE)
	{
		toScratch(scratchpad, inputVector);
		int stride = upsweep(scratchpad);
		T max = upsweepCorrection(scratchpad, tempSums);
		downsweep(scratchpad, tempSums, stride);

		if (maximums != NULL && threadIdx.x == 0)
		{
			maximums[blockIdx.x] = max;
		}

		const int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < ARRAY_SIZE)
		{
			result[tid] = scratchpad[threadIdx.x];
		}
	}
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void scanBlockKernel(T* result, T* inputVector, T* maximums)
{
	__shared__ T scratchpad[BLOCK_SIZE];
	__shared__ T tempSums[BLOCK_SIZE];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < ARRAY_SIZE)
	{
		toScratch(scratchpad, inputVector);
		int stride = upsweep(scratchpad);
		T max = upsweepCorrection(scratchpad, tempSums);
		downsweep(scratchpad, tempSums, stride);
		downsweepCorrection(scratchpad, result, max);

		if (maximums != NULL && threadIdx.x == 0)
		{
			maximums[blockIdx.x] = max;
		}
	}
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
__global__ void scanBlockCorrectionKernel(T* inputVector, const T* blockCorrectionVector)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < ARRAY_SIZE)
	{
		inputVector[tid] += blockCorrectionVector[blockIdx.x];
	}
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
hipError_t getScanCorrectionVector(T* maximums, int blockCount)
{
	// Define block and grid sizes
	dim3 blockSize(blockCount);
	dim3 gridSize(1);

	prescanBlockKernel<T> << <gridSize, blockSize >> > (maximums, maximums, NULL);
	return hipGetLastError();
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
hipError_t gpuScan(T* scanVector, const T* inputVector, bool blockCorrection = false)
{
	T* dev_inputVector = 0, * dev_scan = 0, * dev_max = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on inputVector multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc(&dev_inputVector, ARRAY_SIZE * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_inputVector failed!");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc(&dev_scan, ARRAY_SIZE * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_scan failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_inputVector, inputVector, ARRAY_SIZE * sizeof(T), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_inputVector failed!");
		goto Error;
	}

	// Define block and grid sizes
	dim3 blockSize(BLOCK_SIZE);
	dim3 gridSize((ARRAY_SIZE + blockSize.x - 1) / blockSize.x);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc(&dev_max, gridSize.x * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_max failed!");
		goto Error;
	}

	printf("Launching cuda on grid size %d with block size %d\n", gridSize.x, blockSize.x);
	// Launch inputVector kernel on the GPU with one thread for each element.
	scanBlockKernel << <gridSize, blockSize >> > (dev_scan, dev_inputVector, dev_max);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "scanBlockKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching scanBlockKernel!\n", cudaStatus);
		goto Error;
	}

	if (blockCorrection)
	{
		getScanCorrectionVector(dev_max, gridSize.x);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getScanCorrectionVector launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching getScanCorrectionVector!\n", cudaStatus);
			goto Error;
		}

		scanBlockCorrectionKernel << <gridSize, blockSize >> > (dev_scan, dev_max);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "scanBlockCorrectionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching scanBlockCorrectionKernel!\n", cudaStatus);
			goto Error;
		}
	}

	cudaStatus = hipMemcpy(scanVector, dev_scan, ARRAY_SIZE * sizeof(T), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	hipFree(dev_inputVector); hipFree(dev_scan), hipFree(dev_max);

	return cudaStatus;
}

static void generateRandomVector(int* matrix, int N) {
	for (int i = 0; i < N; ++i) {
		matrix[i] = rand() % 100; // Fill with random values from 0 to 99
	}
}

template<
	typename T,
	typename = typename std::enable_if<std::is_arithmetic<T>::value, T>::type>
bool verify(T* expected, T* actual, int size)
{
	for (int i = 0; i < size; i++)
	{
		if (expected[i] != actual[i])
		{
			return false;
		}
	}

	return true;
}

int main()
{
	int* gpuVector = (int*)malloc(ARRAY_SIZE * sizeof(int));
	int* cpuVector = (int*)malloc(ARRAY_SIZE * sizeof(int));
	int* gpuScanOutput = (int*)malloc(ARRAY_SIZE * sizeof(int));
	int* cpuScanOutput = (int*)malloc(ARRAY_SIZE * sizeof(int));

	assert(gpuVector); assert(cpuVector); assert(gpuScanOutput); assert(cpuScanOutput);

	generateRandomVector(gpuVector, ARRAY_SIZE);
	memcpy(cpuVector, gpuVector, ARRAY_SIZE * sizeof(int));

	// Add vectors in parallel.
	hipError_t cudaStatus = gpuScan(gpuScanOutput, gpuVector, true);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	cpuScan(cpuScanOutput, cpuVector, ARRAY_SIZE);

	if (verify(cpuScanOutput, gpuScanOutput, ARRAY_SIZE))
	{
		printf("OK\n");
	}
	else
	{
		printf("Fail\n");
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
